
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void gpu(int num){
    printf("%d \n", num);
}
int main(){
    for(int i=0;i<N;i++){
        hipStream_t stream;
        hipStreamCreate(&stream);
        gpu<<<1,1, 0, stream>>>(i);
        hipStreamDestroy(stream);
    }
    hipDeviceSynchronize(); // must
}