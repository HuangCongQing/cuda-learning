
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void gpu(int num){
    printf("%d \n", num);
}
int main(){
    for(int i=0;i<N;i++){
        hipStream_t stream;
        hipStreamCreate(&stream); //创建流
        gpu<<<1,1, 0, stream>>>(i); // 作为参数传进核函数
        hipStreamDestroy(stream); // 销毁流
    }
    hipDeviceSynchronize(); // must
}