/*
 * @Description: 矩阵加法  https://www.yuque.com/huangzhongqing/cuda/gnoswcuwb6dgg9ri
 * @Author: HCQ
 * @Company(School): UCAS
 * @Email: 1756260160@qq.com
 * @Date: 2023-03-15 15:28:41
 * @LastEditTime: 2023-03-15 16:01:35
 * @FilePath: /cuda-learning/practice/01vecAdd/02vecAddPro.cu
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
// #include <Eigen/Dense>
// using namespace Eigen;
using namespace std;


// 两个向量加法kernel，grid和block均为一维
__global__ void add(float* x, float * y, float* z, int n)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] + y[i];
    }
}

int main()
{
    int N = 1 << 20;
    int nBytes = N * sizeof(float);

    // 申请托管内存（使用一个托管内存来共同管理host和device中的内存）
    float *x, *y, *z;
    hipMallocManaged((void**)&x, nBytes);
    hipMallocManaged((void**)&y, nBytes);
    hipMallocManaged((void**)&z, nBytes);

    // 初始化数据
    for (int i = 0; i < N; ++i)
    {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    // 定义kernel的执行配置
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    // 执行kernel
    add<<< gridSize, blockSize >>>(x, y, z, N);

    // 同步device 保证结果能正确访问
    hipDeviceSynchronize();
    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(z[i] - 30.0));
    std::cout << "最大误差: " << maxError << std::endl;

    // 释放内存
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}