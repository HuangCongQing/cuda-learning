// 矩阵转置

#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_DIM 32
#define BLOCK_SIZE 8
#define MX 2048
#define MY 2048 

__global__ void transpose(float* odata, float* idata){
    int x = blockIdx.x * TILE_DIM + threadIdx.x; // 一个block里面分TILE_DIM*TILE_DIM，每个TILE_DIM厘米有thread线程数量
    int y = blockIdx.y * TILE_DIM + threadIdx.y;

    int w = gridDim.x * TILE_DIM;
    if(x >= MX || y >=MY) return;
    for(int i = 0; i<TILE_DIM; i += BLOCK_SIZE){ // 一个block是一个矩阵中的一个数
        odata[x * w + y + i] = idata[(y + i) * w + x];
    }
}


// 验证
bool check(float *c_cpu, float * c_gpu){
    for(int r = 0;r < MX; r++){
        for(int c = 0;c < MY; c++){
            if(c_cpu[r * MX +c] != c_gpu[r * MY + c]){
                return false;
            }
        }
    }
    return true;
}

int main(){
    size_t size = MX * MY * sizeof(float);
    float *h_idata, *h_odata, *d_idata, *d_odata, *res; // host device
    hipHostMalloc(&h_idata, size, hipHostMallocDefault);
    hipHostMalloc(&h_odata, size, hipHostMallocDefault); 
    hipHostMalloc(&res, size, hipHostMallocDefault); 
    hipMalloc(&d_idata, size);
    hipMalloc(&d_odata, size);

    // 2 定义threads
    dim3 threads(TILE_DIM, BLOCK_SIZE, 1); // 1维写成3维
    dim3 blocks((MX + TILE_DIM - 1) / TILE_DIM, (MY + TILE_DIM - 1) / TILE_DIM, 1 );
    // c初始化
    for(int i = 0; i < MX;i++){
        for(int j = 0; j < MY; j++){
            h_idata[i * MY + j] = i * MY + j;
            res[i * MY + j] = j * MY + i;
        }
    }
    
    hipMemcpy(d_idata, h_idata, size, hipMemcpyHostToDevice); // cpu->>gpu
    // 多跑几轮
    // for(int i = 0; i < 100; i++)
    transpose<<<blocks, threads>>>(d_odata, d_idata);
    hipMemcpy(h_odata, d_odata, size, hipMemcpyDeviceToHost); // gpu->>cpu
    hipDeviceSynchronize();
    check(res, h_odata) ? printf("ok") : printf("error");

    // end: 释放内存
    hipHostFree(h_idata); // 提前写，防止忘记释放内存
    hipHostFree(h_odata); 
    hipHostFree(res); 
    hipFree(d_idata);  //gpu
    hipFree(d_odata);  //gpu


}